
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void hello() {
    printf("Hello world from device\n");
}

int main() {
    hello<<<1, 1>>>();
    cout << "Hello world from host" << endl;
    hipDeviceSynchronize();
    return 0;
}
